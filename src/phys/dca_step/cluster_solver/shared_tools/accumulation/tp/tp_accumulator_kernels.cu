#include "hip/hip_runtime.h"
// Copyright (C) 2009-2016 ETH Zurich
// Copyright (C) 2007?-2016 Center for Nanophase Materials Sciences, ORNL
// All rights reserved.
//
// See LICENSE.txt for terms of usage.
// See CITATION.txt for citation guidelines if you use this code for scientific publications.
//
// Author: Giovanni Balduzzi (gbalduzz@itp.phys.ethz.ch)
//
// Implements the GPU kernels used by the DFT algorithm.

#include "dca/phys/dca_step/cluster_solver/shared_tools/accumulation/tp/kernels_interface.hpp"

#include <array>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "dca/util/integer_division.hpp"
#include "dca/linalg/util/cast_cuda.hpp"
#include "dca/linalg/util/complex_operators_cuda.cu.hpp"
#include "dca/phys/dca_step/cluster_solver/shared_tools/accumulation/tp/g4_helper.cuh"

namespace dca {
namespace phys {
namespace solver {
namespace accumulator {
namespace details {
namespace global {
// dca::phys::solver::accumulator::details::global::
G4HelperManager helper;
}  // global
// dca::phys::solver::accumulator::details::

using namespace linalg;
using linalg::util::CudaComplex;
using linalg::util::castCudaComplex;

std::array<dim3, 2> getBlockSize(const uint i, const uint j, const uint block_size = 32) {
  const uint n_threads_i = std::min(block_size, i);
  const uint n_threads_j = std::min(block_size, j);
  if (n_threads_i * n_threads_j > 32 * 32)
    throw(std::logic_error("Block size is too big"));

  const uint n_blocks_i = dca::util::ceilDiv(i, n_threads_i);
  const uint n_blocks_j = dca::util::ceilDiv(j, n_threads_j);

  return std::array<dim3, 2>{dim3(n_blocks_i, n_blocks_j), dim3(n_threads_i, n_threads_j)};
}

template <typename Real>
__global__ void computeGSinglebandKernel(CudaComplex<Real>* __restrict__ G, int ldg,
                                         const CudaComplex<Real>* __restrict__ G0, int nk,
                                         int nw_pos, const Real beta) {
  const int n_rows = nk * nw_pos;
  const int n_cols = n_rows * 2;
  const int id_i = blockIdx.x * blockDim.x + threadIdx.x;
  const int id_j = blockIdx.y * blockDim.y + threadIdx.y;
  if (id_i >= n_rows || id_j >= n_cols)
    return;

  auto get_indices = [=](const int id, int& k, int& w) {
    w = id / nk;
    k = id - nk * w;
  };
  int w1, w2, k1, k2;
  get_indices(id_i, k1, w1);
  get_indices(id_j, k2, w2);

  const CudaComplex<Real> G0_w1 = G0[k1 + nk * (w1 + nw_pos)];
  const CudaComplex<Real> G0_w2 = G0[k2 + nk * w2];

  G[id_i + ldg * id_j] *= -G0_w1 * G0_w2;
  if (k1 == k2 && w1 + nw_pos == w2) {
    G[id_i + ldg * id_j] += G0_w1 * beta;
  }
}

template <typename Real>
void computeGSingleband(std::complex<Real>* G, int ldg, const std::complex<Real>* G0, int nk,
                        int nw_pos, const Real beta, hipStream_t stream) {
  const int n_rows = nk * nw_pos;
  auto blocks = getBlockSize(n_rows, n_rows * 2);

  computeGSinglebandKernel<<<blocks[0], blocks[1], 0, stream>>>(
      castCudaComplex(G), ldg, castCudaComplex(G0), nk, nw_pos, beta);
}

template <typename Real>
__global__ void computeGMultibandKernel(CudaComplex<Real>* __restrict__ G, int ldg,
                                        const CudaComplex<Real>* __restrict__ G0, int ldg0, int nb,
                                        int nk, int nw_pos, Real beta) {
  const int id_i = blockIdx.x * blockDim.x + threadIdx.x;
  const int id_j = blockIdx.y * blockDim.y + threadIdx.y;

  assert(id_i < nb * nk * nw_pos);
  assert(id_j < nb * nk * nw_pos * 2);

  const int no = nb * nk;
  auto get_indices = [=](int id, int& b, int& k, int& w) {
    w = id / no;
    id -= w * no;
    k = id / nb;
    b = id - k * nb;
  };
  int w1, w2, k1, k2, b1, b2;
  get_indices(id_i, b1, k1, w1);
  get_indices(id_j, b2, k2, w2);
  w1 += nw_pos;

  // Note: cuda does not support templated shared memory.
  extern __shared__ char shared_mem[];
  CudaComplex<Real>* const M_block = reinterpret_cast<CudaComplex<Real>*>(shared_mem);
  const int local_row_start = (threadIdx.y / nb) * nb;
  const int local_col_start = (threadIdx.x / nb) * nb;
  const int ldm = blockDim.y;
  CudaComplex<Real>* const M = M_block + local_row_start + ldm * local_col_start;

  CudaComplex<Real>& G_val = G[id_i + ldg * id_j];
  M[b1 + ldm * b2] = G_val;
  __syncthreads();

  const CudaComplex<Real>* const G0_w1 = G0 + nb * k1 + no * w1;
  const CudaComplex<Real>* const G0_w2 = G0 + nb * k2 + no * w2;

  G_val.x = G_val.y = 0;
  for (int j = 0; j < nb; ++j) {
    const CudaComplex<Real> G0_w2_val = G0_w2[j + ldg0 * b2];
    for (int i = 0; i < nb; ++i)
      G_val -= G0_w1[b1 + ldg0 * i] * M[i + ldm * j] * G0_w2_val;
  }

  if (G0_w1 == G0_w2)
    G_val += G0_w1[b1 + ldg0 * b2] * beta;
}

template <typename Real>
void computeGMultiband(std::complex<Real>* G, int ldg, const std::complex<Real>* G0, int ldg0,
                       int nb, int nk, int nw_pos, Real beta, hipStream_t stream) {
  const int n_rows = nb * nk * nw_pos;

  auto get_block_width = [nb] {
    if (nb > 16)
      throw(std::logic_error("Too many bands."));
    for (int candidate = 16; candidate > 0; --candidate)
      if (!(candidate % nb))
        return candidate;
    return -1;
  };
  const static int width = get_block_width();

  const static auto blocks = getBlockSize(n_rows, n_rows * 2, width);

  computeGMultibandKernel<<<blocks[0], blocks[1], width * width * sizeof(std::complex<Real>), stream>>>(
      castCudaComplex(G), ldg, castCudaComplex(G0), ldg0, nb, nk, nw_pos, beta);
}

void initializeG4Helpers(int nb, int nk, int nw_pos, int delta_k, int delta_w, const int* add_k,
                         int lda, const int* sub_k, int lds) {
  if (!global::helper.isInitialized())
    global::helper.set(nb, nk, nw_pos, delta_k, delta_w, add_k, lda, sub_k, lds);
  assert(hipPeekAtLastError() == hipSuccess);
}

// Include specializations for each mode
#include "dca/phys/dca_step/cluster_solver/shared_tools/accumulation/tp/modes/particle_particle_up_down.inc"
#include "dca/phys/dca_step/cluster_solver/shared_tools/accumulation/tp/modes/particle_hole_transverse.inc"
#include "dca/phys/dca_step/cluster_solver/shared_tools/accumulation/tp/modes/particle_hole_charge.inc"
#include "dca/phys/dca_step/cluster_solver/shared_tools/accumulation/tp/modes/particle_hole_magnetic.inc"

// Explicit instantiation.
template void computeGSingleband<float>(std::complex<float>* G, int ldg,
                                        const std::complex<float>* G0, int nk, int nw,
                                        const float beta, hipStream_t stream);
template void computeGMultiband<float>(std::complex<float>* G, int ldg,
                                       const std::complex<float>* G0, int ldg0, int nb, int nk,
                                       int nw, float beta, hipStream_t stream);

template void computeGSingleband<double>(std::complex<double>* G, int ldg,
                                         const std::complex<double>* G0, int nk, int nw_pos,
                                         const double beta, hipStream_t stream);
template void computeGMultiband<double>(std::complex<double>* G, int ldg,
                                        const std::complex<double>* G0, int ldg0, int nb, int nk,
                                        int nw_pos, double beta, hipStream_t stream);

}  // details
}  // accumulator
}  // solver
}  // phys
}  // dca
