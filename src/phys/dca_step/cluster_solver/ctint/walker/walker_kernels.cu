#include "hip/hip_runtime.h"
// Copyright (C) 2009-2016 ETH Zurich
// Copyright (C) 2007?-2016 Center for Nanophase Materials Sciences, ORNL
// All rights reserved.
//
// See LICENSE.txt for terms of usage.
// See CITATION.txt for citation guidelines if you use this code for scientific publications.
//
// Author: Giovanni Balduzzi (gbalduzz@itp.phys.ethz.ch)
//
//

#include "dca/phys/dca_step/cluster_solver/ctint/walker/kernels_interface.hpp"

#include <array>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "dca/util/cuda_blocks.hpp"

namespace dca {
namespace phys {
namespace solver {
namespace ctint {
namespace details {

__global__ void setRightSectorToIdKernel(double* m, const int ldm, const int n0, const int n_max) {
  const int i = threadIdx.x + blockDim.x * blockIdx.x;
  const int j = threadIdx.y + blockDim.y * blockIdx.y + n0;

  if (i >= n_max || j >= n_max)
    return;

  m[i + ldm * j] = (i == j) ? 1. : 0.;
}

void setRightSectorToId(double* m, const int ldm, const int n0, const int n_max, hipStream_t stream) {
  auto blocks = dca::util::getBlockSize(n_max, n_max - n0);

  setRightSectorToIdKernel<<<blocks[0], blocks[1], 0, stream>>>(m, ldm, n0, n_max);
}

__global__ void computeGLeftKernel(MatrixView G, const MatrixView M, const double* f, int n_init) {
  const int i = threadIdx.x + blockDim.x * blockIdx.x;
  const int j = threadIdx.y + blockDim.y * blockIdx.y;
  if (i >= G.nrRows() || j >= n_init)
    return;

  G(i, j) = (M(i, j) * f[j] - double(i == j)) / (f[j] - 1);
}

void computeGLeft(MatrixView& G, const MatrixView& M, const double* f, int n_init,
                  hipStream_t stream) {
  if (n_init == 0)
    return;
  const int n = G.nrRows();
  const auto blocks = dca::util::getBlockSize(n, n_init);

  computeGLeftKernel<<<blocks[0], blocks[1], 0, stream>>>(G, M, f, n_init);
}

__global__ void multiplyByFFactorKernel(MatrixView M, const double* f_vals,
                                        const bool inverse_factor, const bool row_factor) {
  const int i = threadIdx.x + blockDim.x * blockIdx.x;
  const int j = threadIdx.y + blockDim.y * blockIdx.y;
  if (i >= M.nrRows() || j >= M.nrCols())
    return;

  double factor;
  if (row_factor)
    factor = -(f_vals[i] - 1.);
  else
    factor = f_vals[j] - 1.;

  if (inverse_factor)
    M(i, j) /= factor;
  else
    M(i, j) *= factor;
}

void multiplyByFFactor(MatrixView& M, const double* f_vals, bool inverse_factor, bool row_factor,
                       hipStream_t stream) {
  if (M.nrCols() == 0 || M.nrRows() == 0)
    return;
  const auto blocks = dca::util::getBlockSize(M.nrRows(), M.nrCols());

  multiplyByFFactorKernel<<<blocks[0], blocks[1], 0, stream>>>(M, f_vals, inverse_factor, row_factor);
}

}  // details
}  // ctint
}  // solver
}  // phys
}  // dca
