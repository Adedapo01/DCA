#include "hip/hip_runtime.h"
// Copyright (C) 2009-2016 ETH Zurich
// Copyright (C) 2007?-2016 Center for Nanophase Materials Sciences, ORNL
// All rights reserved.
//
// See LICENSE.txt for terms of usage.
// See CITATION.txt for citation guidelines if you use this code for scientific publications.
//
// Author: Giovanni Balduzzi (gbalduzz@itp.phys.ethz.ch)
//
//

#include "dca/phys/dca_step/cluster_solver/ctint/walker/kernels_interface.hpp"

#include <array>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "dca/util/integer_division.hpp"

namespace dca {
namespace phys {
namespace solver {
namespace ctint {
namespace details {

std::array<dim3, 2> getBlockSize(const int i, const int j) {
  assert(i > 0 && j > 0);
  const int n_threads_i = std::min(32, i);
  const int n_threads_j = std::min(32, j);
  const int n_blocks_i = util::ceilDiv(i, n_threads_i);
  const int n_blocks_j = util::ceilDiv(j, n_threads_j);

  return std::array<dim3, 2>{dim3(n_blocks_i, n_blocks_j), dim3(n_threads_i, n_threads_j)};
}

__global__ void setRightSectorToIdKernel(double* m, const int ldm, const int n0, const int n_max) {
  const int i = threadIdx.x + blockDim.x * blockIdx.x;
  const int j = threadIdx.y + blockDim.y * blockIdx.y + n0;

  if (i >= n_max || j >= n_max)
    return;

  m[i + ldm * j] = (i == j) ? 1. : 0.;
}

void setRightSectorToId(double* m, const int ldm, const int n0, const int n_max, hipStream_t stream) {
  auto blocks = getBlockSize(n_max, n_max - n0);

  setRightSectorToIdKernel<<<blocks[0], blocks[1], 0, stream>>>(m, ldm, n0, n_max);
}

}  // details
}  // ctint
}  // solver
}  // phys
}  // dca