// Collects all GPU code.

// #define NDEBUG  // This is now added by CMake to the CUDA_NVCC_FLAGS.
// #define DEBUG_CUDA
// #define hipDeviceScheduleBlockingSync 0x04

#include "hip/hip_runtime.h"

#include "dca/util/integer_division.hpp"
#include "dca/linalg/util/error_cuda.hpp"
#include "dca/linalg/util/stream_functions.hpp"

#include "phys_library/DCA+_step/cluster_solver/cluster_solver_mc_ctaux/ctaux_walker/ctaux_walker_routines_GPU.cu.h"
#include "phys_library/DCA+_step/cluster_solver/cluster_solver_mc_ctaux/ctaux_walker/ctaux_walker_tools/ctaux_G0_matrix_routines/ctaux_G0_matrix_routines_GPU.cu.h"
#include "phys_library/DCA+_step/cluster_solver/cluster_solver_mc_ctaux/ctaux_walker/ctaux_walker_tools/ctaux_G_matrix_routines/ctaux_G_matrix_routines_GPU.cu.h"
#include "phys_library/DCA+_step/cluster_solver/cluster_solver_mc_ctaux/ctaux_walker/ctaux_walker_tools/ctaux_N_matrix_routines/ctaux_N_matrix_routines_GPU.cu.h"
