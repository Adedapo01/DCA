#include "hip/hip_runtime.h"
// Copyright (C) 2009-2016 ETH Zurich
// Copyright (C) 2007?-2016 Center for Nanophase Materials Sciences, ORNL
// All rights reserved.
//
// See LICENSE.txt for terms of usage.
// See CITATION.txt for citation guidelines if you use this code for scientific publications.
//
// Author: Giovanni Balduzzi (gbalduzz@itp.phys.ethz.ch)
//
// Implements the GPU kernels used by the SpaceTransform2DGpu class.

#include "dca/math/function_transform/special_transforms/kernels_interface.hpp"

#include <array>

#include "dca/util/integer_division.hpp"
#include "dca/linalg/util/cast_cuda.hpp"

namespace dca {
namespace math {
namespace transform {
namespace details {
// dca::math::transform::details::

using linalg::util::CudaComplex;
using linalg::util::castCudaComplex;

std::array<dim3, 2> getBlockSize(const int i, const int j) {
  const int n_threads_i = std::min(32, i);
  const int n_threads_j = std::min(32, j);
  const int n_blocks_i = util::ceilDiv(i, n_threads_i);
  const int n_blocks_j = util::ceilDiv(j, n_threads_j);

  return std::array<dim3, 2>{dim3(n_blocks_i, n_blocks_j), dim3(n_threads_i, n_threads_j)};
}

template <typename Real>
__global__ void rearrangeResultKernel(const CudaComplex<Real>* in, const int ldi,
                                      CudaComplex<Real>* out, const int ldo, const int nb,
                                      const int nk, const int nw) {
  const int id_i = blockIdx.x * blockDim.x + threadIdx.x;
  const int id_j = blockIdx.y * blockDim.y + threadIdx.y;

  const int cols = nb * nk * nw;
  const int rows = cols / 2;
  if (id_i >= rows || id_j >= cols)
    return;

  const int no = nb * nk;
  auto get_indices = [nk, no](int id, int& b, int& k, int& w) {
    w = id / no;
    id -= w * no;
    b = id / nk;
    k = id - b * nk;
  };
  int w1, w2, b1, b2, k1, k2;

  get_indices(id_i, b1, k1, w1);
  get_indices(id_j, b2, k2, w2);

  const int out_i = b1 + nb * k1 + no * w1;
  const int out_j = b2 + nb * k2 + no * w2;

  out[out_i + ldo * out_j] = in[id_i + ldi * id_j];
}

template <typename Real>
void rearrangeResult(const std::complex<Real>* in, const int ldi, std::complex<Real>* out,
                     const int ldo, const int nb, const int nk, const int nw,
                     const hipStream_t stream) {
  const int size = nk * nb * nw;
  auto const blocks = getBlockSize(size / 2, size);

  rearrangeResultKernel<Real><<<blocks[0], blocks[1], 0, stream>>>(
      castCudaComplex(in), ldi, castCudaComplex(out), ldo, nb, nk, nw);
}

// Explicit instantiation.
template void rearrangeResult<double>(const std::complex<double>* in, const int ldi,
                                      std::complex<double>* out, const int ldo, const int nb,
                                      const int nk, const int nw, hipStream_t stream);
template void rearrangeResult<float>(const std::complex<float>* in, const int ldi,
                                     std::complex<float>* out, const int ldo, const int nb,
                                     const int nk, const int nw, hipStream_t stream);

}  // details
}  // transform
}  // math
}  // dca
